#include "hip/hip_runtime.h"
﻿#include<stdio.h>

#include "hip/hip_runtime.h"
#include ""

//Warp details 
//Some threads are remaining idle
//Tried to use cout , but didn't work. we should use C I/O functions 
__global__ void print_details_of_warps()
{
	int gid = blockIdx.y * gridDim.x * blockDim.x +
		blockIdx.x * blockDim.x + threadIdx.x;
	int warp_id = threadIdx.x / 32;

	int gbid = blockIdx.x * gridDim.x + blockIdx.x;

	printf("tid : %d, bid.x: %d, bid.y: %d, gid: %d, warp_id: %d, gbid: %d\n",
		threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
}

int main(int argc, char** argv)
{
	dim3 block_size(42);
	dim3 grid_size(2, 2);

	print_details_of_warps << <grid_size, block_size>> > ();

	hipDeviceSynchronize();
	return EXIT_SUCCESS;
}