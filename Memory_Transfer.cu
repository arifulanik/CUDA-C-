#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "" 

#include <stdio.h> 
#include <stdlib.h>
#include<cstring>
#include<time.h>

//S1L12:  Memory transfer between host and device
__global__ void mem_trs_test(int * input) 
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	printf("tid : %d, gid : %d, value : %d \n", threadIdx.x, gid, input[gid] );
}

__global__ void mem_trs_test2(int* input, int size) //More optimize . 
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	if(gid<size)
		printf("tid : %d, gid : %d, value : %d \n", threadIdx.x, gid, input[gid]);
	//if we don't give this condition still the other 10 threads will try to acess the array and it'll find the value 0
}

int main()
{
	int size = 150;
	int byte_size = size * sizeof(int);

	int* h_input;
	h_input = (int*) malloc(byte_size);

	//time to randomly initialize the array
	time_t t;
	srand((unsigned)time(&t));
	for (int i = -0; i < size; i++)
	{
		h_input[i] = (int)(rand() & 0xff);
	}
	
	int * d_input;
	hipMalloc((void**) &d_input, byte_size);
	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);


	dim3 block(32);
	dim3 grid(5);
	//mem_trs_test << <grid, block >> > (d_input);
	mem_trs_test2 << <grid, block >> > (d_input, size);
	

	//thread block size affects the performance. 
	//usually we keep the thread block size to multiplication of 32

	hipDeviceSynchronize();

	//Reclaim memory
	hipFree(d_input);
	free(h_input);

	hipDeviceReset();
	return 0;
}